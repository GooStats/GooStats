/*****************************************************************************/
// Author: Xuefeng Ding <xuefeng.ding.physics@gmail.com>
// Insitute: Gran Sasso Science Institute, L'Aquila, 67100, Italy
// Date: 2018 April 7th
// Version: v1.0
// Description: GooStats, a statistical analysis toolkit that runs on GPU.
//
// All rights reserved. 2018 copyrighted.
/*****************************************************************************/
#include <cstdio>

#include "GPUManager.h"
#include "GooStatsException.h"
bool GPUManager::preinit() {
  if (!report()) {
    throw GooStatsException("Cannot find a free GPU!");
  }
  return true;
}
#ifdef __HIPCC__
bool GPUManager::report(bool siliently) const {
  int devicesCount(1);
  hipGetDeviceCount(&devicesCount);
  if (devicesCount > 1000) {
    printf(
        "number of devices larger than 1000. this is suspicious. Please modify the code here or login to the GPU "
        "node.");
    return false;
  }
  for (int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex) {
    printf("Checking GPU [%d]/[%d]\n", deviceIndex, devicesCount);
    if (report(deviceIndex, siliently))
      return true;
  }
  return false;
#else
bool GPUManager::report(bool) const {
  return true;
#endif
}
#ifdef __HIPCC__
bool GPUManager::report(int gpu_id, bool siliently) const {
  hipSetDevice(gpu_id);
  hipError_t cuda_status;
  //// show memory usage of GPU
  //size_t free_byte ;
  //size_t total_byte ;
  //cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
  //if ( hipSuccess != cuda_status ){
  //  printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
  //  return false;
  //}
  //if(!siliently) {
  //  double free_db = (double)free_byte ;
  //  double total_db = (double)total_byte ;
  //  double used_db = total_db - free_db ;
  //  printf("GPU memory usage: used = %lf, free = %lf MB, total = %lf MB\n",
  //      used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
  //}
  int device;
  cuda_status = hipGetDevice(&device);
  if (hipSuccess != cuda_status) {
    printf("Error: hipGetDevice fails, %s \n", hipGetErrorString(cuda_status));
    return false;
  }
  hipDeviceProp_t prop;
  cuda_status = hipGetDeviceProperties(&prop, device);
  if (hipSuccess != cuda_status) {
    printf("Error: hipGetDeviceProperties fails, %s \n", hipGetErrorString(cuda_status));
    return false;
  }
  int *test;
  cuda_status = hipMalloc((void **)&test, sizeof(int));
  if (hipSuccess != cuda_status) {
    printf("Error: hipMalloc fails, %s \n", hipGetErrorString(cuda_status));
    return false;
  }
  hipFree(test);
  if (!siliently) {
    printf("Running on [%s] with compute capability of %d.%d\n", prop.name, prop.major, prop.minor);
  }
//  if( !(prop.major>3 || prop.major==3 && prop.minor>=5) )
//    return false;
#else
bool GPUManager::report(int, bool) const {
#endif
  return true;
}
